#include "hip/hip_runtime.h"
#include "definitions.cuh"

//Performs CFD calculation on global memory. This code does not use any advance optimization technique on GPU
// But still acheives many fold performance gain
__global__ void calculateCFD_V1( float* input,  float* output, unsigned int Ni, unsigned int Nj, 
								   float h)
{
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x; // Y - ID
	unsigned int j = blockDim.y * blockIdx.y + threadIdx.y; // X - ID

	unsigned int iPrev = i-1; // Previous Y element
	unsigned int iNext = i+1; // Next Y element

	unsigned int jPrev = j-1; //Previous X element
	unsigned int jNext = j+1; // Next X element


	unsigned int index = i * Nj + j;

	if( i > 0 && j > 0 && i < (Ni-1) && j <(Nj-1))
		output[index] = 0.25f * (input[iPrev * Nj + j] + input[iNext* Nj + j] + input[i * Nj+ jPrev] 
			+ input[i* Nj + jNext] - 4*h*h);
}
