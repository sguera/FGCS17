#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include "definitions.cuh"
#include <time.h>
#include <stdio.h>
#include "util.h"
#include "dim_input.h"

//Number of elements on which to perform CFD
unsigned int Ni = PAR_I; //512; // Y elements
unsigned int Nj = PAR_J; //512; // X elements
unsigned int nIterations = PAR_ITER; //10000; // No Of Iterations

int main( int argc, char** argv ) {

    //Variables for graph
	char metric[] = "ms";
	
	//Variables for Timing
	float gpuTime;

	// CPU and GPU Pointers ( d_XX : refers to pointer pointing to GPU memory. This is just a convention)
	float *t = NULL, *t_prev = NULL;
	float *d_t = NULL,*d_t_prev= NULL;

    if ( ( Ni % THREADS_PER_BLOCK_Y != 0 )  || ( Nj % THREADS_PER_BLOCK_X != 0 ) ) {
        fprintf( stderr, "Please specify Ni & Nj as multiple of 16 !!!!" );
        exit( 0 );
    }

	printf("\n Ni= %d, Nj=%d nIteration=%d",Ni,Nj,nIterations);
	
	//unsigned int size = Ni * Nj * sizeof(float);

	if ( !initializeCPU( &t, &t_prev ) ) {
		printf( "\n Error in allocating memory on CPU!!!" );
		unInitializeCPU( &t, &t_prev );
		return 0;
	}

	if ( !initializeGPU( &d_t, &d_t_prev ) ) {
		printf( "\n Error in allocating memory on GPU!!!" );
		unInitializeCPU( &t, &t_prev );
		unInitializeGPU( &d_t, &d_t_prev );
		return 0;
	}

	//Perform CFD on GPU
	if ( !performGPUCFD( d_t,d_t_prev, t, t_prev, &gpuTime ) ) {
		printf( "\n GPU Kernel failed !!!" );
		unInitializeCPU( &t, &t_prev );
		unInitializeGPU( &d_t, &d_t_prev );
		return 0;
	}

    /* information read by the tool for printing graphs*/
    printMetric( metric, gpuTime );
    
	unInitializeCPU( &t, &t_prev );
	unInitializeGPU( &d_t, &d_t_prev );
	
	return EXIT_SUCCESS;

}

int initializeCPU( float **t, float **t_prev ) {
	*t = (float*) calloc( Ni*Nj, sizeof( float ) );
	*t_prev = (float*) calloc( Ni*Nj, sizeof( float ) );

	if ( (*t) == NULL || (*t_prev) == NULL )
		return 0;
	else
		return 1;
}

void unInitializeCPU( float **t, float **t_prev ) {
	if ( (*t) != NULL )
		free( *t );
	if ( (*t_prev) != NULL )
		free( *t_prev );
}

int initializeGPU( float **d_t, float **d_t_prev ) {

	unsigned int size = Ni * Nj * sizeof(float);

	// Choose which GPU to run on, change this on a multi-GPU system.
    hipError_t cudaStatus = hipSetDevice(0);
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?" );
        return 0;
    }
	 // Allocate GPU buffers.
    cudaStatus = hipMalloc( (void**) &(*d_t), size );
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipMalloc failed!" );
        return 0;
    }

	 // Allocate GPU buffers   .
    cudaStatus = hipMalloc( (void**) &(*d_t_prev), size );
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipMalloc failed!" );
        return 0;
    }

	 // Memset GPU buffers
    cudaStatus = hipMemset( (*d_t), 0, size );
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipMemset failed!" );
        return 0;
    }

	// Memset GPU buffers
    cudaStatus = hipMemset( (*d_t_prev), 0, size );
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipMemset failed!" );
        return 0;
    }

	return 1;
}

void unInitializeGPU( float **d_t, float **d_t_prev ) {
	hipError_t cudaStatus;

	if ( (*d_t) != NULL )
        cudaStatus = hipFree( (*d_t) );
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipFree failed!" );
        return;
    }

	if ( (*d_t_prev) != NULL )
        cudaStatus = hipFree( (*d_t_prev) );
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipFree failed!" );
        return;
    }

	cudaStatus = hipDeviceReset();
    if ( cudaStatus != hipSuccess ) {
        fprintf( stderr, "hipDeviceReset failed!" );
        return;
    }
}

int performGPUCFD( float *d_t, float *d_t_prev, float *t, float *t_prev, float *gpuTime ) {

	float h,x,y;
	const char *str = (char*) malloc( 1024 ); // To store error string
	
	//Decide how many blocks per thread and how many blocks per grid
	dim3 dimBlock( THREADS_PER_BLOCK_X, THREADS_PER_BLOCK_Y );
	dim3 dimGrid( Nj/dimBlock.x, Ni/dimBlock.y );
	
	h = 1.0f/(Ni-1);
	memset( t_prev, 0, sizeof(float) * Ni * Nj );
  
	for ( unsigned int i=0; i<Ni; i++ ) {
		x = i*h;
		t_prev[i*Nj+0] = x*x;
		t_prev[i*Nj+(Nj-1)] = x*x + 1.0f;
	}

	for ( unsigned int j=0; j<Nj; j++ ) {
		y = j*h;
		t_prev[0*Nj+j] = y*y;
		t_prev[((Ni-1) * Nj) + j] = 1.0f + y*y;
	}

	//Copy data to device
	hipMemcpy( d_t_prev, t_prev, sizeof(float) * Ni * Nj , hipMemcpyHostToDevice );

	//Insert event to calculate time
	hipEvent_t start, stop;
	hipEventCreate( &start );
	hipEventCreate( &stop );

	//This calls Version 1 of kernel which uses Global memory
	hipEventRecord( start, 0 );
	
	for ( unsigned int k=0; k<nIterations; k++)	{
		// Launch a kernel on the GPU with one thread for each element.
		calculateCFD_V1<<<dimGrid,dimBlock>>>( d_t_prev,d_t, Ni, Nj, h );
		
		float* pingPong = d_t_prev;
		d_t_prev = d_t;
		d_t = pingPong;
	}
	hipEventRecord( stop, 0 );
	hipEventSynchronize( stop );

	float elapsedTime;
	hipEventElapsedTime( &elapsedTime, start, stop );
	//printf( "\n GPU Time:: %f ms", elapsedTime );

	*gpuTime = elapsedTime;

	hipError_t cudaStatus = hipMemcpy( t, d_t_prev, sizeof(float) * Ni * Nj , hipMemcpyDeviceToHost );
	if ( cudaStatus != hipSuccess ) {
		fprintf( stderr, "hipMemcpy failed!" );
		str = hipGetErrorString( cudaStatus );
		fprintf( stderr, "CUDA Error!:: %s\n", str );
		return 0;
	}
	
	return 1;
}

